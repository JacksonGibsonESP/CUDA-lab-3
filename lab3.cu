#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
//#include <chrono>

using namespace std;

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define LIMIT(a,b,c) (MIN(MAX((a),(b)),(c)))

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
				    }													\
} while (0)

__global__ void kernel_histogram(unsigned char *src, int length, unsigned int *histogram)
{
	__shared__ unsigned int local_histogram[256];
	if (threadIdx.x == 0)
	{
		memset(&local_histogram, 0, sizeof(unsigned int) * 256);
	}
	__syncthreads();
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	while (tid < length)
	{
		atomicAdd(&local_histogram[src[tid]], 1);
		tid += gridDim.x * blockDim.x;
	}
	__syncthreads();
	atomicAdd(&histogram[threadIdx.x], local_histogram[threadIdx.x]);
}

__global__ void kernel_scan(unsigned int *histogram)
{
	__shared__ unsigned int local_histogram[256];
	if (threadIdx.x == 0)
	{
		memcpy(&local_histogram, histogram, sizeof(unsigned int) * 256);
	}
	__syncthreads();
	int pow = 2;
	while (pow <= 256)
	{
		if (threadIdx.x * pow + pow - 1 < 256)
		{
			local_histogram[threadIdx.x * pow + pow - 1] += local_histogram[threadIdx.x * pow + pow - pow / 2 - 1];
		}
		__syncthreads();
		pow *= 2;
	}
	if (threadIdx.x == 0)
	{
		local_histogram[255] = 0;
	}
	__syncthreads();

	pow = 256;
	while (pow > 1)
	{
		if (threadIdx.x * pow + pow - 1 < 256)
		{
			unsigned int sum = local_histogram[threadIdx.x * pow + pow - 1] + local_histogram[threadIdx.x * pow + pow - pow / 2 - 1];
			local_histogram[threadIdx.x * pow + pow - pow / 2 - 1] = local_histogram[threadIdx.x * pow + pow - 1];
			local_histogram[threadIdx.x * pow + pow - 1] = sum;
		}
		__syncthreads();
		pow /= 2;
	}
	histogram[threadIdx.x] += local_histogram[threadIdx.x];
}

int main()
{
	int length = 0;
	fread(&length, sizeof(int), 1, stdin); 
	
	if (length != 0)
	{
		unsigned char *src = (unsigned char *)malloc(sizeof(unsigned char) * length);
		unsigned char *dst = (unsigned char *)malloc(sizeof(unsigned char) * length);
		fread(src, sizeof(unsigned char), length, stdin);

		unsigned char *src_dev;
		CSC(hipMalloc(&src_dev, sizeof(unsigned char) * length));
		CSC(hipMemcpy(src_dev, src, sizeof(unsigned char) * length, hipMemcpyHostToDevice));

		unsigned int *histogram_dev;
		CSC(hipMalloc(&histogram_dev, sizeof(unsigned int) * 256));
		hipMemset(histogram_dev, 0, sizeof(unsigned int) * 256);
		//auto start_time = chrono::high_resolution_clock::now();
		kernel_histogram << <32, 256 >> >(src_dev, length, histogram_dev);

		kernel_scan << <1, 256 >> >(histogram_dev);

		unsigned int histogram[256];
		CSC(hipMemcpy(&histogram, histogram_dev, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost));

		for (int i = 0; i < length; i++)
		{
			dst[histogram[src[i]] - 1] = src[i];
			histogram[src[i]]--;
		}
		//auto end_time = chrono::high_resolution_clock::now();
		//cout << '\n' << chrono::duration_cast<chrono::milliseconds>(end_time - start_time).count() << "ms\n";
		fwrite(dst, sizeof(unsigned char), length, stdout);

		free(dst);
		free(src);
	}
	return 0;
}